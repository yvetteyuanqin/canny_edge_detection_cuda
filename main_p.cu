/**
*  Sequential canny detection
*
*
*/
//Yuan Qin
//Lai Man Tang

//Mar.6th.2018
#include <boost/gil/rgb.hpp>
#include <boost/gil/extension/io/png_dynamic_io.hpp>
#include <boost/fusion/view.hpp>
#include <boost/fusion/include/view.hpp>
#include <stdint.h>
#include <vector>
#include <iostream>
#include "canny_p.h"
#include "timer.h"
#define HEIGHT 512
#define WIDTH 512

using namespace std;

int main() {

struct stopwatch_t* timer = NULL;
long double t_gaussian, t_gradient,t_nms,t_thres;
/* initialize timer */
stopwatch_init ();
timer = stopwatch_create ();
stopwatch_start (timer);
//    t_gaussian= stopwatch_stop (timer);
//    cout<< "Time to execute gaussian:"<< t_gaussian<<endl;


//string filename = argv[1]
using namespace boost::gil;
gray8_image_t img(HEIGHT,WIDTH);
png_read_image("001.png", img);
//png_write_view("testimg.png",const_view(img));


// Get a raw pointer to the gray buffer
unsigned char * buf = &view(img)[0][0];//new unsigned char[w * h];
gray8_pixel_t **h_imgbuff = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*WIDTH);
for(int i = 0; i < WIDTH; i++)
{
*(h_imgbuff+i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*HEIGHT);
}

/*put the image pixel into the gray buffer*/
gray8_view_t srcview = interleaved_view(img.width(), img.height(), (gray8_pixel_t*)buf, view(img).pixels().row_size());

for (int i = 0; i < HEIGHT; ++i) {
for (int j = 0; j < WIDTH; ++j) {
h_imgbuff[i][j]=srcview(j, i);
}
}

/*canny_sequential*/
gray8_pixel_t **h_newImage = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
for(int i = 0; i < WIDTH; i++)
{
*(h_newImage+i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
for(int j = 0;j<HEIGHT;j++) h_newImage[i][j]=0;
}

gray8_pixel_t **d_imgbuff;
gray8_pixel_t **d_newImage;
hipMalloc(&d_imgbuff,sizeof(gray8_pixel_t*)*HEIGHT);
hipMalloc(&d_newImage,sizeof(gray8_pixel_t*)*HEIGHT);
for(int i = 0; i < WIDTH; i++)
{
hipMalloc(&d_imgbuff[i],sizeof(gray8_pixel_t)*WIDTH);
hipMalloc(&d_newImage+i,sizeof(gray8_pixel_t)*WIDTH);
}
//memcopy
hipMemcpy2D(d_imgbuff, sizeof(gray8_pixel_t)*WIDTH, h_imgbuff, sizeof(gray8_pixel_t) * WIDTH, sizeof(gray8_pixel_t) *WIDTH, HEIGHT, hipMemcpyHostToDevice);
//    for(int i = 0; i < WIDTH; i++)
//    {
//        hipMemcpy(d_imgbuff, h_imgbuff, sizeof(gray8_pixel_t) * HEIGHT, hipMemcpyHostToDevice);
//    }
//



/*apply gaussian filter*/
cout << "enter gaussian filter" << endl;
int numBlocks = 1;
dim3 threadsPerBlock(HEIGHT, WIDTH);
stopwatch_start (timer);
gaussian_filter<<<numBlocks,threadsPerBlock>>>(d_newImage,d_imgbuff,WIDTH, HEIGHT);
t_gaussian= stopwatch_stop (timer);

//MEMCOPY BACK TO HOST
hipMemcpy2D(h_newImage, sizeof(gray8_pixel_t)*WIDTH, d_newImage, sizeof(gray8_pixel_t) * WIDTH, sizeof(gray8_pixel_t) *WIDTH, HEIGHT, hipMemcpyDeviceToHost);

//free device mem

for(int i = 0; i < WIDTH; i++)
{
hipFree(d_imgbuff+i);
hipFree(d_newImage+i);
}
hipFree(d_newImage);
hipFree(d_imgbuff);

cout<< "Time to execute gaussian:"<< t_gaussian<<endl;
cout << "finished." << endl;


//    gray8_pixel_t **gradientImg = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
//    for (int i = 0; i < WIDTH; i++)
//    {
//        *(gradientImg + i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
//        for (int j = 0; j<HEIGHT; j++) gradientImg[i][j] = 0;
//    }

//    gray8_pixel_t **deltaX = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
//    gray8_pixel_t **deltaY = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
//
//    for (int i = 0; i < WIDTH; i++)
//    {
//        *(deltaX + i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
//        *(deltaY + i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
//    }
//
//    /*Gradient*/
//    cout << "enter gradient filter" << endl;
//    stopwatch_start (timer);
//     gradient(gradientImg, newImage, WIDTH, HEIGHT,deltaX,deltaY);
//    t_gradient= stopwatch_stop (timer);
//
//    cout<< "Time to execute gradient:"<< t_gradient<<endl;
//
//    cout << "finished." << endl;
//
//
//    gray8_pixel_t **NMSImg = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
//    for (int i = 0; i < WIDTH; i++)
//    {
//        *(NMSImg + i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
//        for (int j = 0; j<HEIGHT; j++) NMSImg[i][j] = 0;
//    }
//
//    /*non maximum suppression*/
//    cout << "enter suppression filter" << endl;
//    stopwatch_start (timer);
//        suppress(NMSImg,gradientImg,WIDTH, HEIGHT,deltaX,deltaY);
//    t_nms= stopwatch_stop (timer);
//
//    cout<< "Time to execute nms:"<< t_nms<<endl;
//
//    cout << "finished." << endl;
//
//
//    gray8_pixel_t **thresImg = (gray8_pixel_t**)malloc(sizeof(gray8_pixel_t*)*HEIGHT);
//    for (int i = 0; i < WIDTH; i++)
//    {
//        *(thresImg + i) = (gray8_pixel_t*)malloc(sizeof(gray8_pixel_t)*WIDTH);
//        for (int j = 0; j<HEIGHT; j++) thresImg[i][j] = 0;
//    }
//
//    /*non hysterious thresholding*/
//    cout << "enter hysterious" << endl;
//    unsigned char hi = 0xFC;
//    unsigned char lo = 0xC0;
//    stopwatch_start (timer);
//    apply_hysteresis(thresImg,NMSImg, hi, lo, WIDTH,HEIGHT);
//    t_thres= stopwatch_stop (timer);
//
//    cout<< "Time to execute thresholded:"<< t_thres<<endl;


cout << "finished." << endl;


/*revert image to check*/
gray8_image_t img2(HEIGHT,WIDTH);
const View &img_view = view(img2);

cout << "create image view" << endl;
for (int i = 0; i < HEIGHT; ++i) {
for (int j = 0; j < WIDTH; ++j) {
img_view(j, i) = h_newImage[i][j];
}
}
cout << "finished." << endl;

png_write_view("testimg.png", const_view(img2));


return 0;

}
