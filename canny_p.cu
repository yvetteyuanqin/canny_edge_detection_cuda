#include "hip/hip_runtime.h"
#include <stdint.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <stdio.h>
#include "canny_p.h"
//#include "timer.h"

using namespace std;

/*create a gaussian filter*/
//__global__
//double** createKernel(int height, int width, double sigma)
//{
//Matrix kernel(height, Array(width));
//
//double **d_kernel;
//hipMalloc(&d_kernel, sizeof(unsigned char*)*height);
//for (int i = 0; i < width; i++)
//{
//hipMalloc(&d_kernel[i], sizeof(unsigned char)*width);
//}
//
//double sum=0.0;
//int i,j;
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
//sum += d_kernel[i][j];
//}
//}
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] /= sum;
//}
//}
//
//return d_kernel;
//}

/*Step 1 blur the image to reduce noice*/
__global__
void gaussian_filter(unsigned char **newImage, unsigned char **in_pixels, int width, int height)
{
	// create kernel

	int hi = 5;
	int wd = 5;
	__shared__ double filter[5][5];

	//=(double **)malloc(sizeof(double*)*hi);
	//for (int i = 0; i < wd; i++)
	//{
	//*(filter+i)=(double *)malloc(sizeof(double)*wd);
	//}

	/*allocate newimage*/
	int i = threadIdx.x;
	int j = threadIdx.y;


	double sum = 0.0;
	printf("creating filter");
	double sigma = 10.0;
	for (int h = 0; i<hi; h++) {
		for (int w = 0; j<wd; w++) {
			filter[h][w] = exp(-(h*h + w*w) / (2 * sigma*sigma)) / (2 * M_PI*sigma*sigma);
			sum += filter[h][w];
		}
	}

	__syncthreads();

	for (int i = 0; i<hi; i++) {
		for (int j = 0; j<wd; j++) {
			filter[i][j] /= sum;
		}
	}

	__syncthreads();

	printf("finish filter");

	//start filtering
	//double** filter = createKernel(5, 5, 10.0);
	int filterHeight = 5;
	int filterWidth = 5;
	int newImageHeight = height - filterHeight;
	int newImageWidth = width - filterWidth;



	int h, w;
	printf("apply gaussian filter");	
	//
	//        for (i=0 ; i<newImageHeight ; i++) {
	//            for (j=0 ; j<newImageWidth ; j++) {
	for (h = i; h<i + filterHeight; h++) {
		for (w = j; w<j + filterWidth; w++) {
			newImage[i][j] = newImage[i][j] + filter[h - i][w - j] * in_pixels[h][w];
		}
	}
	__syncthreads();
	//            }
	//        }
	//__syncthreads();
	printf("finish gaussian filter");

}
__global__
void gradient(unsigned char **newImage, unsigned char **in_pixels, int width, int height,
	unsigned char **deltaX, unsigned char **deltaY)
{


	// compute delta X ***************************
	// deltaX = f(x+1) - f(x-1)
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (j == 0) deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j];
			else if (j == width - 1)deltaX[i][j] = in_pixels[i][j] - in_pixels[i][j - 1];
			else deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j - 1];
		}
	}
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			if (i == 0) deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i][j];
			else if (i == height - 1)deltaY[i][j] = in_pixels[i][j] - in_pixels[i - 1][j];
			else deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i - 1][j];
		}
	}
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			newImage[i][j] = (unsigned char)(sqrt((double)deltaX[i][j] * deltaX[i][j] +
				(double)deltaY[i][j] * deltaY[i][j]) + 0.5);
		}
	}

}

__global__
void suppress(unsigned char **newImage, unsigned char **mag, int width, int height,
	unsigned char **deltaX, unsigned char **deltaY)
{

	float alpha;
	float mag1, mag2;
	// put zero all boundaries of image
	// TOP edge line of the image
	for (int k = 0; k < height; ++k) {
		newImage[height - 1][k] = 0;
		newImage[0][k] = 0;
		newImage[k][0] = 0;
		newImage[k][width - 1] = 0;
	}


	// skip boundaries of image
	// start and stop 1 pixel inner pixels from boundaries
	for (unsigned i = 1; i < height - 1; i++)
	{
		for (unsigned j = 1; j < width - 1; j++)
		{
			// if magnitude = 0, no edge
			if (mag[i][j] == 0) newImage[i][j] = 0;//suppressed
			else {
				if (deltaX[i][j] >= 0)
				{
					if (deltaY[i][j] >= 0)  // dx >= 0, dy >= 0
					{
						if ((deltaX[i][j] - deltaY[i][j]) >= 0)       // direction 1 (SEE, South-East-East)
						{
							alpha = (float)deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
							mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
						}
						else                                // direction 2 (SSE)
						{
							alpha = (float)deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];

						}
					}

					else  // dx >= 0, dy < 0
					{
						if ((deltaX[i][j] + deltaY[i][j]) >= 0)    // direction 8 (NEE)
						{
							alpha = (float)-deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
							mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];

						}
						else                                // direction 7 (NNE)
						{

							alpha = (float)deltaX[i][j] / -deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];

						}
					}
				}

				else
				{
					if (deltaY[i][j] >= 0) // dx < 0, dy >= 0
					{
						if ((deltaX[i][j] - deltaY[i][j]) >= 0)    // direction 3 (SSW)
						{
							alpha = (float)-deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];
						}
						else                                // direction 4 (SWW)
						{
							alpha = (float)deltaY[i][j] / -deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
						}
					}

					else // dx < 0, dy < 0
					{
						if ((-deltaX[i][j] + deltaY[i][j]) >= 0)   // direction 5 (NWW)
						{
							alpha = (float)deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
							mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
						}
						else                                // direction 6 (NNW)
						{
							alpha = (float)deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];
							mag2 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
						}
					}
				}

				// non-maximal suppression
				// compare mag1, mag2 and mag[t]
				// if mag[t] is smaller than one of the neighbours then suppress it
				if ((mag[i][j] < mag1) || (mag[i][j] < mag2))
					newImage[i][j] = 0;//SUPRRESSED
				else
				{
					newImage[i][j] = mag[i][j];
				}

			}
		}
	}
}
__global__
void apply_hysteresis(unsigned char **out_pixels, unsigned char **in_pixels, unsigned  char t_high, unsigned  char t_low, int width, int height)
{
	/* skip first and last rows and columns, since we'll check them as surrounding neighbors of
	* the adjacent rows and columns */

	for (unsigned i = 1; i < height - 1; i++) {
		for (unsigned j = 1; j < width - 1; j++) {
			/* if our input is above the high threshold and the output hasn't already marked it as an edge */
			if (out_pixels[i][j] != 0xFF) {
				if (in_pixels[i][j] > t_high) {
					/* mark as strong edge */
					out_pixels[i][j] = 0xff;

					/* check 8 immediately surrounding neighbors
					* if any of the neighbors are above the low threshold, preserve edge */
					trace_immed_neighbors(out_pixels, in_pixels, i, j, t_low);
				}
				else {
					out_pixels[i][j] = 0x00;
				}
			}
		}
	}
}
__device__
void trace_immed_neighbors(unsigned char **out_pixels, unsigned char **in_pixels, unsigned i, unsigned j, unsigned char t_low)
{

	unsigned char m_edge = 255;
	/* directions representing indices of neighbors */

	if (((unsigned char)in_pixels[i - 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j - 1] != m_edge)) {
		out_pixels[i - 1][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i - 1][j] >= t_low) && ((unsigned char)out_pixels[i - 1][j] != m_edge)) {
		out_pixels[i - 1][j] = m_edge;
	}
	if (((unsigned char)in_pixels[i - 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j + 1] != m_edge)) {
		out_pixels[i - 1][j + 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i][j - 1] >= t_low) && ((unsigned char)out_pixels[i][j - 1] != m_edge)) {
		out_pixels[i][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i][j + 1] >= t_low) && ((unsigned char)out_pixels[i][j + 1] != m_edge)) {
		out_pixels[i][j + 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j - 1] != m_edge)) {
		out_pixels[i + 1][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j] >= t_low) && ((unsigned char)out_pixels[i + 1][j] != m_edge)) {
		out_pixels[i + 1][j] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j + 1] != m_edge)) {
		out_pixels[i + 1][j + 1] = m_edge;
	}
}

void edge_detector(unsigned char** h_newImg, unsigned char** h_imgbuff, int WIDTH, int HEIGHT) {

	/* initialize timer */
	//struct stopwatch_t* timer = NULL;
	//long double t_gaussian, t_gradient, t_nms, t_thres;
	//stopwatch_init();
	//timer = stopwatch_create();

	unsigned char **d_imgbuff;
	unsigned char **d_newImage;
	cout<<"hipMalloc"<<endl;
	hipMalloc((void**)&d_imgbuff, sizeof(unsigned char*)*HEIGHT);
	hipMalloc((void**)&d_newImage, sizeof(unsigned char*)*HEIGHT);
	printf("cuda1D finish");
	for (int i = 0; i < WIDTH; i++)
	{
		hipMalloc(&d_imgbuff[i], sizeof(unsigned char)*WIDTH);
		hipMalloc(&d_newImage[i], sizeof(unsigned char)*WIDTH);
	}
	//memcopy
	printf("cuda2D finish");
	hipMemcpy2D(d_imgbuff, sizeof(unsigned char)*WIDTH, h_imgbuff, sizeof(unsigned char) * WIDTH, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyHostToDevice);
	cout<<"hipMalloc finished"<<endl;

	/*apply gaussian filter*/
	cout << "enter gaussian filter" << endl;
	int numBlocks = 1;
	dim3 threadsPerBlock(HEIGHT, WIDTH);
	//stopwatch_start(timer);
	gaussian_filter << <numBlocks, threadsPerBlock >> >(d_newImage, d_imgbuff, WIDTH, HEIGHT);
	//t_gaussian = stopwatch_stop(timer);

	//MEMCOPY BACK TO HOST
	hipMemcpy2D(h_newImg, sizeof(unsigned char)*WIDTH, d_newImage, sizeof(unsigned char) * WIDTH, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyDeviceToHost);

	//free device mem

	cout<<"Free cuda"<<endl;
	for (int i = 0; i < WIDTH; i++)
	{
		hipFree(d_imgbuff + i);
		hipFree(d_newImage + i);
	}
	hipFree(d_newImage);
	hipFree(d_imgbuff);

	//cout << "Time to execute gaussian:" << t_gaussian << endl;
	cout << "finished." << endl;


}

