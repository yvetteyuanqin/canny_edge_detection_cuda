#include "hip/hip_runtime.h"
#include <stdint.h>
#include <math.h>
#include <iostream>
#include <vector>
#include "canny_p.h"
#include "timer.h"



using namespace std;

/*create a gaussian filter*/
__device__
Matrix createKernel(int height, int width, double sigma)
{
Matrix kernel(height, Array(width));
double sum=0.0;
int i,j;

for (i=0 ; i<height ; i++) {
for (j=0 ; j<width ; j++) {
kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
sum += kernel[i][j];
}
}

for (i=0 ; i<height ; i++) {
for (j=0 ; j<width ; j++) {
kernel[i][j] /= sum;
}
}

return kernel;
}

/*Step 1 blur the image to reduce noice*/
__global__
void gaussian_filter(unsigned char **newImage,unsigned char **in_pixels,int width, int height)
{

Matrix filter = createKernel(5, 5, 10.0);
int filterHeight = filter.size();
int filterWidth = filter[0].size();
int newImageHeight = height-filterHeight;
int newImageWidth = width-filterWidth;



int h,w;
/*allocate newimage*/
int i = threadIdx.x;
int j = threadIdx.y;
//
//        for (i=0 ; i<newImageHeight ; i++) {
//            for (j=0 ; j<newImageWidth ; j++) {
for (h=i ; h<i+filterHeight ; h++) {
for (w=j ; w<j+filterWidth ; w++) {
newImage[i][j] = newImage[i][j] +filter[h-i][w-j]*in_pixels[h][w];
}
}
__syncthreads();
//            }
//        }
//__syncthreads();

}
__global__
void gradient(unsigned char **newImage, unsigned char **in_pixels, int width, int height,
unsigned char **deltaX, unsigned char **deltaY)
{


// compute delta X ***************************
// deltaX = f(x+1) - f(x-1)
for (int i = 0; i < height; i++) {
for (int j = 0; j < width; j++){
if (j == 0) deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j];
else if (j == width - 1)deltaX[i][j] = in_pixels[i][j] - in_pixels[i][j - 1];
else deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j - 1];
}
}
for (int i = 0; i < width; i++) {
for (int j = 0; j < height; j++) {
if (i == 0) deltaY[i][j] = in_pixels[i+1][j] - in_pixels[i][j];
else if (i == height - 1)deltaY[i][j] = in_pixels[i][j] - in_pixels[i-1][j];
else deltaY[i][j] = in_pixels[i+1][j] - in_pixels[i-1][j];
}
}
for (int i = 0; i < width; i++) {
for (int j = 0; j < height; j++) {
newImage[i][j] = (unsigned char)(sqrt((double)deltaX[i][j] * deltaX[i][j] +
(double)deltaY[i][j] * deltaY[i][j]) + 0.5);
}
}

}

__global__
void suppress(unsigned char **newImage, unsigned char **mag, int width, int height,
unsigned char **deltaX, unsigned char **deltaY)
{

float alpha;
float mag1, mag2;
// put zero all boundaries of image
// TOP edge line of the image
for (int k = 0; k < height; ++k) {
newImage[height - 1][k] = 0;
newImage[0][k] = 0;
newImage[k][0] = 0;
newImage[k][width - 1]= 0;
}


// skip boundaries of image
// start and stop 1 pixel inner pixels from boundaries
for(unsigned i = 1; i < height-1; i++)
{
for(unsigned j = 1; j < width-1; j++)
{
// if magnitude = 0, no edge
if(mag[i][j] == 0) newImage[i][j] = 0;//suppressed
else{
if(deltaX[i][j] >= 0)
{
if(deltaY[i][j] >= 0)  // dx >= 0, dy >= 0
{
if((deltaX[i][j] - deltaY[i][j]) >= 0)       // direction 1 (SEE, South-East-East)
{
alpha = (float)deltaY[i][j] / deltaX[i][j];
mag1 = (1-alpha)*mag[i][j+1] + alpha*mag[i+1][j+1];
mag2 = (1-alpha)*mag[i][j-1] + alpha*mag[i-1][j-1];
}
else                                // direction 2 (SSE)
{
alpha = (float)deltaX[i][j] / deltaY[i][j];
mag1 = (1-alpha)*mag[i+1][j] + alpha*mag[i+1][j+1];
mag2 = (1-alpha)*mag[i-1][j] + alpha*mag[i-1][j-1];

}
}

else  // dx >= 0, dy < 0
{
if((deltaX[i][j] + deltaY[i][j]) >= 0)    // direction 8 (NEE)
{
alpha = (float)-deltaY[i][j] / deltaX[i][j];
mag1 = (1-alpha)*mag[i][j+1] + alpha*mag[i-1][j+1];
mag2 = (1-alpha)*mag[i][j-1] + alpha*mag[i+1][j-1];

}
else                                // direction 7 (NNE)
{

alpha = (float)deltaX[i][j] / -deltaY[i][j];
mag1 = (1-alpha)*mag[i+1][j] + alpha*mag[i+1][j-1];
mag2 = (1-alpha)*mag[i-1][j] + alpha*mag[i-1][j+1];

}
}
}

else
{
if(deltaY[i][j] >= 0) // dx < 0, dy >= 0
{
if((deltaX[i][j] - deltaY[i][j]) >= 0)    // direction 3 (SSW)
{
alpha = (float)-deltaX[i][j] / deltaY[i][j];
mag1 = (1-alpha)*mag[i+1][j] + alpha*mag[i+1][j-1];
mag2 = (1-alpha)*mag[i-1][j] + alpha*mag[i-1][j+1];
}
else                                // direction 4 (SWW)
{
alpha = (float)deltaY[i][j] / -deltaX[i][j];
mag1 = (1-alpha)*mag[i][j-1] + alpha*mag[i+1][j-1];
mag2 = (1-alpha)*mag[i][j+1] + alpha*mag[i-1][j+1];
}
}

else // dx < 0, dy < 0
{
if((-deltaX[i][j] + deltaY[i][j]) >= 0)   // direction 5 (NWW)
{
alpha = (float)deltaY[i][j] / deltaX[i][j];
mag1 = (1-alpha)*mag[i][j-1] + alpha*mag[i-1][j-1];
mag2 = (1-alpha)*mag[i][j+1] + alpha*mag[i+1][j+1];
}
else                                // direction 6 (NNW)
{
alpha = (float)deltaX[i][j] / deltaY[i][j];
mag1 = (1-alpha)*mag[i-1][j] + alpha*mag[i-1][j-1];
mag2 = (1-alpha)*mag[i+1][j] + alpha*mag[i+1][j+1];
}
}
}

// non-maximal suppression
// compare mag1, mag2 and mag[t]
// if mag[t] is smaller than one of the neighbours then suppress it
if((mag[i][j] < mag1) || (mag[i][j] < mag2))
newImage[i][j] = 0;//SUPRRESSED
else
{
newImage[i][j] = mag[i][j];
}

}
}
}
}
__global__
void apply_hysteresis(unsigned char **out_pixels, unsigned char **in_pixels, unsigned  char t_high, unsigned  char t_low, int width,int height)
{
/* skip first and last rows and columns, since we'll check them as surrounding neighbors of
* the adjacent rows and columns */

for(unsigned i = 1; i < height-1; i++) {
for(unsigned j = 1; j < width-1; j++) {
/* if our input is above the high threshold and the output hasn't already marked it as an edge */
if (out_pixels[i][j] != 0xFF) {
if (in_pixels[i][j] > t_high) {
/* mark as strong edge */
out_pixels[i][j] = 0xff;

/* check 8 immediately surrounding neighbors
* if any of the neighbors are above the low threshold, preserve edge */
trace_immed_neighbors(out_pixels, in_pixels, i,j, t_low);
} else {
out_pixels[i][j] = 0x00;
}
}
}
}
}
__device__
void trace_immed_neighbors(unsigned char **out_pixels, unsigned char **in_pixels, unsigned i, unsigned j, unsigned char t_low)
{

unsigned char m_edge= 255;
/* directions representing indices of neighbors */

if (((unsigned char)in_pixels[i-1][j-1] >= t_low) && ((unsigned char)out_pixels[i-1][j-1] != m_edge)) {
out_pixels[i-1][j-1] = m_edge;
}
if (((unsigned char)in_pixels[i-1][j] >= t_low) && ((unsigned char)out_pixels[i-1][j] != m_edge)) {
out_pixels[i-1][j] = m_edge;
}
if (((unsigned char)in_pixels[i-1][j+1] >= t_low) && ((unsigned char)out_pixels[i-1][j+1] != m_edge)) {
out_pixels[i-1][j+1] = m_edge;
}
if (((unsigned char)in_pixels[i][j-1] >= t_low) && ((unsigned char)out_pixels[i][j-1]  != m_edge)) {
out_pixels[i][j-1]  = m_edge;
}
if (((unsigned char)in_pixels[i][j+1] >= t_low) && ((unsigned char)out_pixels[i][j+1] != m_edge)) {
out_pixels[i][j+1] = m_edge;
}
if (((unsigned char)in_pixels[i+1][j-1] >= t_low) && ((unsigned char)out_pixels[i+1][j-1] != m_edge)) {
out_pixels[i+1][j-1] = m_edge;
}
if (((unsigned char)in_pixels[i+1][j] >= t_low) && ((unsigned char)out_pixels[i+1][j] != m_edge)) {
out_pixels[i+1][j] = m_edge;
}
if (((unsigned char)in_pixels[i+1][j+1] >= t_low) && ((unsigned char)out_pixels[i+1][j+1] != m_edge)) {
out_pixels[i+1][j+1] = m_edge;
}
}

void edge_detector(unsigned char** h_newImg, unsigned char** h_imgbuff, int WIDTH, int HEIGHT){

/* initialize timer */
struct stopwatch_t* timer = NULL;
long double t_gaussian, t_gradient, t_nms, t_thres;
stopwatch_init();
timer = stopwatch_create();

unsigned char **d_imgbuff;
unsigned char **d_newImage;
hipMalloc(&d_imgbuff, sizeof(unsigned char*)*HEIGHT);
hipMalloc(&d_newImage, sizeof(unsigned char*)*HEIGHT);
for (int i = 0; i < WIDTH; i++)
{
hipMalloc(&d_imgbuff[i], sizeof(unsigned char)*WIDTH);
hipMalloc(&d_newImage + i, sizeof(unsigned char)*WIDTH);
}
//memcopy
hipMemcpy2D(d_imgbuff, sizeof(unsigned char)*WIDTH, h_imgbuff, sizeof(unsigned char) * WIDTH, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyHostToDevice);


/*apply gaussian filter*/
cout << "enter gaussian filter" << endl;
int numBlocks = 1;
dim3 threadsPerBlock(HEIGHT, WIDTH);
stopwatch_start(timer);
gaussian_filter <<<numBlocks, threadsPerBlock >>>(d_newImage, d_imgbuff, WIDTH, HEIGHT);
t_gaussian = stopwatch_stop(timer);

//MEMCOPY BACK TO HOST
hipMemcpy2D(h_newImg, sizeof(unsigned char)*WIDTH, d_newImage, sizeof(unsigned char) * WIDTH, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyDeviceToHost);

//free device mem


for (int i = 0; i < WIDTH; i++)
{
hipFree(d_imgbuff + i);
hipFree(d_newImage + i);
}
hipFree(d_newImage);
hipFree(d_imgbuff);

cout << "Time to execute gaussian:" << t_gaussian << endl;
cout << "finished." << endl;


}

