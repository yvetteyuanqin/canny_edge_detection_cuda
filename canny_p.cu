#include "hip/hip_runtime.h"
#include <stdint.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <stdio.h>
#include "canny_p_cu.h"
//#include "timer.h"

using namespace std;

/*create a gaussian filter*/
//__global__
//double** createKernel(int height, int width, double sigma)
//{
//Matrix kernel(height, Array(width));
//
//double **d_kernel;
//hipMalloc(&d_kernel, sizeof(unsigned char*)*height);
//for (int i = 0; i < width; i++)
//{
//hipMalloc(&d_kernel[i], sizeof(unsigned char)*width);
//}
//
//double sum=0.0;
//int i,j;
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
//sum += d_kernel[i][j];
//}
//}
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] /= sum;
//}
//}
//
//return d_kernel;
//}

/*Step 1 blur the image to reduce noice*/
__global__
void gaussian_filter(unsigned char **newImagetmp, unsigned char **in_pixelstmp, const int width, const int height, double filter[25])
{
	// create kernel

	//int hi = 5;
	//int wd = 5;
	/*allocate newimage*/
	
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int threadID = blockId * (blockDim.x * blockDim.y) + ( threadIdx.y * blockDim.x) + threadIdx.x;
	
	int i = threadID /512;
	int j = threadID % 512;

	printf("TS [%d][%d] \n",i ,j );








	unsigned char in_pixels = 0;
	if(in_pixelstmp[i][j] == NULL)
		printf("Error in [%d][%d]", i, j);
	else in_pixels = in_pixelstmp[i][j];


	__syncthreads();
	//        filter[0][0] = 1 / 273, filter[0][1] = 4 / 273, filter[0][2] = 7 / 273, filter[0][3] = 4 / 273, filter[0][4] = 1 / 273,
	//        filter[1][0] = 4 / 273, filter[1][1] = 16 / 273, filter[1][2] = 26 / 273, filter[1][3] = 16 / 273, filter[1][4] = 4 / 273,
	//        filter[2][0] = 7 / 273, filter[2][1] = 26 / 273, filter[2][2] = 41 / 273, filter[2][3] = 26 / 273, filter[2][4] = 7 / 273,
	//        filter[3][0] = 4 / 273, filter[3][1] = 16 / 273, filter[3][2] = 26 / 273, filter[3][3] = 16 / 273, filter[3][4] = 4 / 273,
	//        filter[4][0] = 1 / 273, filter[4][1] = 4 / 273, filter[4][2] = 7 / 273, filter[4][3] = 4 / 273, filter[4][4] = 1 / 273;
	//

	/*flattening */
	//__shared__ unsigned char newImage[width][height];
	//unsigned char in_pixels[512][512];
	//unsigned char newImage[512][512];

	//printf("filter created");


	/*
	if (i == 0 && j==0){
	double sum = 0.0;

	printf("creating filter");
	double sigma = 10.0;
	for (int h = 0; h<hi; h++) {
	for (int w = 0; w<wd; w++) {
	filter[h][w] = exp(-(h*h + w*w) / (2 * sigma*sigma)) / (2 * M_PI*sigma*sigma);
	sum += filter[h][w];
	}
	}


	for (int h = 0; h<hi; h++) {
	for (int w = 0; w<wd; w++) {
	filter[h][w] /= sum;
	}
	}
	}
	__syncthreads();*/



	//start filtering
	//double** filter = createKernel(5, 5, 10.0);


	
	//printf("apply gaussian filter");
	//
	//        for (i=0 ; i<newImageHeight ; i++) {
	//            for (j=0 ; j<newImageWidth ; j++) {
	//if (i < newImageHeight && j < newImageWidth) {
	if (i < height && j < width) {
		//unsigned char* row = (unsigned char*)((unsigned char*)in_pixelstmp + i * pitch);
		//unsigned char* row = (unsigned char*)((unsigned char*)in_pixelstmp + i * width);
		

		//newImagetmp[i*width+j] = 0;
		
		unsigned char pvalue = 0;
		for (int h = 0; h < 25; h++) {
				//newImagetmp[i*width+j] = newImagetmp[i*width+j] + filter[h - i][w - j] * in_pixels;
				pvalue = pvalue + filter[h] * in_pixels;
			
			
		}
		if(in_pixelstmp[i][j] == NULL)
				printf("Error out [%d][%d]\n", i, j);
		else in_pixelstmp[i][j] = pvalue;
	}else printf("TT [%d][%d] \n",i,j);
	__syncthreads();

}


__global__
void gradient(unsigned char **newImage, unsigned char **in_pixels, int width, int height,
	unsigned char **deltaX, unsigned char **deltaY)
{


	// compute delta X ***************************
	// deltaX = f(x+1) - f(x-1)
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (j == 0) deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j];
			else if (j == width - 1)deltaX[i][j] = in_pixels[i][j] - in_pixels[i][j - 1];
			else deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j - 1];
		}
	}
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			if (i == 0) deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i][j];
			else if (i == height - 1)deltaY[i][j] = in_pixels[i][j] - in_pixels[i - 1][j];
			else deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i - 1][j];
		}
	}
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			newImage[i][j] = (unsigned char)(sqrt((double)deltaX[i][j] * deltaX[i][j] +
				(double)deltaY[i][j] * deltaY[i][j]) + 0.5);
		}
	}

}

__global__
void suppress(unsigned char **newImage, unsigned char **mag, int width, int height,
	unsigned char **deltaX, unsigned char **deltaY)
{

	float alpha;
	float mag1, mag2;
	// put zero all boundaries of image
	// TOP edge line of the image
	for (int k = 0; k < height; ++k) {
		newImage[height - 1][k] = 0;
		newImage[0][k] = 0;
		newImage[k][0] = 0;
		newImage[k][width - 1] = 0;
	}


	// skip boundaries of image
	// start and stop 1 pixel inner pixels from boundaries
	for (unsigned i = 1; i < height - 1; i++)
	{
		for (unsigned j = 1; j < width - 1; j++)
		{
			// if magnitude = 0, no edge
			if (mag[i][j] == 0) newImage[i][j] = 0;//suppressed
			else {
				if (deltaX[i][j] >= 0)
				{
					if (deltaY[i][j] >= 0)  // dx >= 0, dy >= 0
					{
						if ((deltaX[i][j] - deltaY[i][j]) >= 0)       // direction 1 (SEE, South-East-East)
						{
							alpha = (float)deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
							mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
						}
						else                                // direction 2 (SSE)
						{
							alpha = (float)deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];

						}
					}

					else  // dx >= 0, dy < 0
					{
						if ((deltaX[i][j] + deltaY[i][j]) >= 0)    // direction 8 (NEE)
						{
							alpha = (float)-deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
							mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];

						}
						else                                // direction 7 (NNE)
						{

							alpha = (float)deltaX[i][j] / -deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];

						}
					}
				}

				else
				{
					if (deltaY[i][j] >= 0) // dx < 0, dy >= 0
					{
						if ((deltaX[i][j] - deltaY[i][j]) >= 0)    // direction 3 (SSW)
						{
							alpha = (float)-deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];
						}
						else                                // direction 4 (SWW)
						{
							alpha = (float)deltaY[i][j] / -deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];
							mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
						}
					}

					else // dx < 0, dy < 0
					{
						if ((-deltaX[i][j] + deltaY[i][j]) >= 0)   // direction 5 (NWW)
						{
							alpha = (float)deltaY[i][j] / deltaX[i][j];
							mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
							mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
						}
						else                                // direction 6 (NNW)
						{
							alpha = (float)deltaX[i][j] / deltaY[i][j];
							mag1 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];
							mag2 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
						}
					}
				}

				// non-maximal suppression
				// compare mag1, mag2 and mag[t]
				// if mag[t] is smaller than one of the neighbours then suppress it
				if ((mag[i][j] < mag1) || (mag[i][j] < mag2))
					newImage[i][j] = 0;//SUPRRESSED
				else
				{
					newImage[i][j] = mag[i][j];
				}

			}
		}
	}
}
__global__
void apply_hysteresis(unsigned char **out_pixels, unsigned char **in_pixels, unsigned  char t_high, unsigned  char t_low, int width, int height)
{
	/* skip first and last rows and columns, since we'll check them as surrounding neighbors of
	* the adjacent rows and columns */

	for (unsigned i = 1; i < height - 1; i++) {
		for (unsigned j = 1; j < width - 1; j++) {
			/* if our input is above the high threshold and the output hasn't already marked it as an edge */
			if (out_pixels[i][j] != 0xFF) {
				if (in_pixels[i][j] > t_high) {
					/* mark as strong edge */
					out_pixels[i][j] = 0xff;

					/* check 8 immediately surrounding neighbors
					* if any of the neighbors are above the low threshold, preserve edge */
					trace_immed_neighbors(out_pixels, in_pixels, i, j, t_low);
				}
				else {
					out_pixels[i][j] = 0x00;
				}
			}
		}
	}
}
__device__
void trace_immed_neighbors(unsigned char **out_pixels, unsigned char **in_pixels, unsigned i, unsigned j, unsigned char t_low)
{

	unsigned char m_edge = 255;
	/* directions representing indices of neighbors */

	if (((unsigned char)in_pixels[i - 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j - 1] != m_edge)) {
		out_pixels[i - 1][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i - 1][j] >= t_low) && ((unsigned char)out_pixels[i - 1][j] != m_edge)) {
		out_pixels[i - 1][j] = m_edge;
	}
	if (((unsigned char)in_pixels[i - 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j + 1] != m_edge)) {
		out_pixels[i - 1][j + 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i][j - 1] >= t_low) && ((unsigned char)out_pixels[i][j - 1] != m_edge)) {
		out_pixels[i][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i][j + 1] >= t_low) && ((unsigned char)out_pixels[i][j + 1] != m_edge)) {
		out_pixels[i][j + 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j - 1] != m_edge)) {
		out_pixels[i + 1][j - 1] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j] >= t_low) && ((unsigned char)out_pixels[i + 1][j] != m_edge)) {
		out_pixels[i + 1][j] = m_edge;
	}
	if (((unsigned char)in_pixels[i + 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j + 1] != m_edge)) {
		out_pixels[i + 1][j + 1] = m_edge;
	}
}

void edge_detector(unsigned char** h_newImg, unsigned char** h_imgbuff, const int WIDTH, const int HEIGHT) {

	/* initialize timer */
	//struct stopwatch_t* timer = NULL;
	//long double t_gaussian, t_gradient, t_nms, t_thres;
	//stopwatch_init();
	//timer = stopwatch_create();

	double filter[25] = { 1 / 273,4 / 273,7 / 273,4 / 273,1 / 273 ,
	 4 / 273,16 / 273,26 / 273,16 / 273,4 / 273 ,
	 7 / 273,26 / 273,41 / 273,26 / 273,7 / 273 ,
	 4 / 273,16 / 273,26 / 273,16 / 273,4 / 273 ,
	 1 / 273,4 / 273,7 / 273,4 / 273,1 / 273  };

	unsigned char **	d_imgbuff;
	unsigned char *		d_imgtemp[WIDTH];
	unsigned char **	d_newImage;
	unsigned char *		d_newimgtemp[WIDTH];
	double d_filter[25];
	cout << "hipMalloc" << endl;

	hipError_t err = hipMalloc((void**)&d_imgbuff, sizeof(unsigned char*)*HEIGHT);

	if (err == 0)	cout << "cuda1D d_imgbuff finish" << endl;
	else cout << "Error :" << err << endl;

	err = hipMalloc((void**)&d_newImage, sizeof(unsigned char*)*HEIGHT);
	if (err == 0)	cout << "cuda1D d_newImage finish" << endl;
	else cout << "Error :" << err << endl;

	cout << "cuda1D finish" << endl;
	for (int i = 0; i < HEIGHT; i++)
	{
		err = hipMalloc((void **)&d_imgtemp[i], sizeof(unsigned char)*WIDTH);
		if (err != hipSuccess) cout << "Error :" << err << " i = " << i << endl;

		err = hipMalloc((void**)&d_newimgtemp[i], sizeof(unsigned char)*WIDTH);
		if (err != hipSuccess) cout << "Error :" << err << " i = " << i << endl;
	}

	//Malloc filter
	err = hipMalloc((void**)&d_filter, sizeof(double)*25);
	if (err == 0)	cout << "cuda1D filter finish" << endl;
	else cout << "Error :" << err << endl;


	//memcopy
	err = hipMemcpy(d_filter, filter,sizeof(unsigned char*)*WIDTH, hipMemcpyHostToDevice);
	if (err != hipSuccess) cout << "Error :" << err << endl;



	err = hipMemcpy(d_imgbuff, d_imgtemp,sizeof(unsigned char*)*WIDTH, hipMemcpyHostToDevice);
	if (err != hipSuccess) cout << "Error :" << err << endl;

	err = hipMemcpy(d_newImage, d_newimgtemp, sizeof(unsigned char*)*WIDTH, hipMemcpyHostToDevice);
	if (err != hipSuccess) cout << "Error :" << err << endl;

	cout << "cuda2D finish" << endl;

	for (int i = 0; i < HEIGHT; i++)
	{
		err = hipMemcpy(d_imgtemp[i], h_imgbuff[i], sizeof(unsigned char)*WIDTH, hipMemcpyHostToDevice);
		if (err != hipSuccess) cout << "Error h_imgtemp :" << err << " i = " << i << endl;

		err = hipMemcpy(d_newimgtemp[i], h_newImg[i], sizeof(unsigned char)*WIDTH, hipMemcpyHostToDevice);
		if (err != hipSuccess) cout << "Error h_newImg :" << err << " i = " << i << endl;


	}

	cout << "hipMalloc finished" << endl;



	
	/*apply gaussian filter*/
	cout << "enter gaussian filter" << endl;
	dim3 threadsPerBlock(4,4);
	dim3 numBlocks (HEIGHT/threadsPerBlock.x, WIDTH/threadsPerBlock.y);
	//stopwatch_start(timer);
	//gaussian_filter << <numBlocks, threadsPerBlock >> >(d_newImage, d_imgbuff, WIDTH, HEIGHT, filter);
	
	err = hipDeviceSynchronize();
	if (err != hipSuccess) cout << "Error hipDeviceSynchronize :" << err << endl;
	
	err = hipDeviceSynchronize();
	if (err != hipSuccess) cout << "Error hipDeviceSynchronize :" << err << endl;

	
	gaussian_filter << <numBlocks, threadsPerBlock >> >(d_newImage, d_imgbuff, WIDTH, HEIGHT, d_filter);
	
	err = hipDeviceSynchronize();
	if (err != hipSuccess) cout << "Error hipDeviceSynchronize :" << err << endl;
	
	err = hipDeviceSynchronize();
	if (err != hipSuccess) cout << "Error hipDeviceSynchronize :" << err << endl;

	//t_gaussian = stopwatch_stop(timer);

	//MEMCOPY BACK TO HOST
	//    err = hipMemcpy2D(h_newImg, WIDTH* sizeof(unsigned char), d_newImage, pitch2, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyDeviceToHost);
	for (int i = 0; i < HEIGHT; i++)
	{

		err = hipMemcpy(h_newImg[i], d_imgbuff[i], sizeof(unsigned char)*WIDTH, hipMemcpyDeviceToHost);
		if (err != hipSuccess) cout << "Error h_newimgtemp :" << err << " i = " << i << endl;
	}

	cout << "MEMCOPY BACK TO HOST finished" << endl;

	//free device mem

	cout << "Free cuda" << endl;
	for (int i = 0; i < WIDTH; i++)
	{
		hipFree(d_imgbuff + i);
		hipFree(d_newImage + i);
	}
	hipFree(d_newImage);
	hipFree(d_imgbuff);

	//cout << "Time to execute gaussian:" << t_gaussian << endl;
	cout << "finished." << endl;


}


