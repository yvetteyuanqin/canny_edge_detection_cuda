#include "hip/hip_runtime.h"
#include <stdint.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <stdio.h>
#include "canny_p_cu.h"
//#include "timer.h"

using namespace std;

/*create a gaussian filter*/
//__global__
//double** createKernel(int height, int width, double sigma)
//{
//Matrix kernel(height, Array(width));
//
//double **d_kernel;
//hipMalloc(&d_kernel, sizeof(unsigned char*)*height);
//for (int i = 0; i < width; i++)
//{
//hipMalloc(&d_kernel[i], sizeof(unsigned char)*width);
//}
//
//double sum=0.0;
//int i,j;
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] = exp(-(i*i+j*j)/(2*sigma*sigma))/(2*M_PI*sigma*sigma);
//sum += d_kernel[i][j];
//}
//}
//
//for (i=0 ; i<height ; i++) {
//for (j=0 ; j<width ; j++) {
//d_kernel[i][j] /= sum;
//}
//}
//
//return d_kernel;
//}

/*Step 1 blur the image to reduce noice*/
__global__
void gaussian_filter(unsigned char *newImagetmp, unsigned char *in_pixelstmp,const int width,const int height, size_t pitch)
{
// create kernel

//int hi = 5;
//int wd = 5;
/*allocate newimage*/
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;

double filter[5][5];

if(i == 0 && j ==0){
filter[0][0] = 1 / 273, filter[0][1] = 4 / 273, filter[0][2] = 7 / 273, filter[0][3] = 4 / 273, filter[0][4] = 1 / 273,
filter[1][0] = 4 / 273, filter[1][1] = 16 / 273, filter[1][2] = 26 / 273, filter[1][3] = 16 / 273, filter[1][4] = 4 / 273,
filter[2][0] = 7 / 273, filter[2][1] = 26 / 273, filter[2][2] = 41 / 273, filter[2][3] = 26 / 273, filter[2][4] = 7 / 273,
filter[3][0] = 4 / 273, filter[3][1] = 16 / 273, filter[3][2] = 26 / 273, filter[3][3] = 16 / 273, filter[3][4] = 4 / 273,
filter[4][0] = 1 / 273, filter[4][1] = 4 / 273, filter[4][2] = 7 / 273, filter[4][3] = 4 / 273, filter[4][4] = 1 / 273;
}

__syncthreads();

/*flattening */
//__shared__ unsigned char newImage[width][height];
unsigned char in_pixels[512][512];
unsigned char newImage[512][512];

printf("shared memory created");


/*
if (i == 0 && j==0){
double sum = 0.0;

printf("creating filter");
double sigma = 10.0;
for (int h = 0; h<hi; h++) {
for (int w = 0; w<wd; w++) {
filter[h][w] = exp(-(h*h + w*w) / (2 * sigma*sigma)) / (2 * M_PI*sigma*sigma);
sum += filter[h][w];
}
}


for (int h = 0; h<hi; h++) {
for (int w = 0; w<wd; w++) {
filter[h][w] /= sum;
}
}
}
__syncthreads();*/

printf("finish filter");

//start filtering
//double** filter = createKernel(5, 5, 10.0);
int filterHeight = 5;
int filterWidth = 5;
int newImageHeight = height - filterHeight;
int newImageWidth = width - filterWidth;



int h, w;
printf("apply gaussian filter");
//
//        for (i=0 ; i<newImageHeight ; i++) {
//            for (j=0 ; j<newImageWidth ; j++) {
if (i < newImageHeight && j < newImageWidth) {

unsigned char* row = (unsigned char*)((unsigned char*)in_pixelstmp + i * pitch);
in_pixels[i][j] = row[j];
newImage[i][j] = 0;
__syncthreads();

//        for (h = i; h<i + filterHeight; h++) {
//            for (w = j; w<j + filterWidth; w++) {
//                newImage[i][j] = newImage[i][j] + filter[h - i][w - j] * in_pixels[h][w];
//            }
//        }
//newImagetmp [i*width+j] = newImage[i][j];

__syncthreads();
}
//            }
//        }
//__syncthreads();





printf("finish gaussian filter");

}
__global__
void gradient(unsigned char **newImage, unsigned char **in_pixels, int width, int height,
unsigned char **deltaX, unsigned char **deltaY)
{


// compute delta X ***************************
// deltaX = f(x+1) - f(x-1)
for (int i = 0; i < height; i++) {
for (int j = 0; j < width; j++) {
if (j == 0) deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j];
else if (j == width - 1)deltaX[i][j] = in_pixels[i][j] - in_pixels[i][j - 1];
else deltaX[i][j] = in_pixels[i][j + 1] - in_pixels[i][j - 1];
}
}
for (int i = 0; i < width; i++) {
for (int j = 0; j < height; j++) {
if (i == 0) deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i][j];
else if (i == height - 1)deltaY[i][j] = in_pixels[i][j] - in_pixels[i - 1][j];
else deltaY[i][j] = in_pixels[i + 1][j] - in_pixels[i - 1][j];
}
}
for (int i = 0; i < width; i++) {
for (int j = 0; j < height; j++) {
newImage[i][j] = (unsigned char)(sqrt((double)deltaX[i][j] * deltaX[i][j] +
(double)deltaY[i][j] * deltaY[i][j]) + 0.5);
}
}

}

__global__
void suppress(unsigned char **newImage, unsigned char **mag, int width, int height,
unsigned char **deltaX, unsigned char **deltaY)
{

float alpha;
float mag1, mag2;
// put zero all boundaries of image
// TOP edge line of the image
for (int k = 0; k < height; ++k) {
newImage[height - 1][k] = 0;
newImage[0][k] = 0;
newImage[k][0] = 0;
newImage[k][width - 1] = 0;
}


// skip boundaries of image
// start and stop 1 pixel inner pixels from boundaries
for (unsigned i = 1; i < height - 1; i++)
{
for (unsigned j = 1; j < width - 1; j++)
{
// if magnitude = 0, no edge
if (mag[i][j] == 0) newImage[i][j] = 0;//suppressed
else {
if (deltaX[i][j] >= 0)
{
if (deltaY[i][j] >= 0)  // dx >= 0, dy >= 0
{
if ((deltaX[i][j] - deltaY[i][j]) >= 0)       // direction 1 (SEE, South-East-East)
{
alpha = (float)deltaY[i][j] / deltaX[i][j];
mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
}
else                                // direction 2 (SSE)
{
alpha = (float)deltaX[i][j] / deltaY[i][j];
mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];

}
}

else  // dx >= 0, dy < 0
{
if ((deltaX[i][j] + deltaY[i][j]) >= 0)    // direction 8 (NEE)
{
alpha = (float)-deltaY[i][j] / deltaX[i][j];
mag1 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
mag2 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];

}
else                                // direction 7 (NNE)
{

alpha = (float)deltaX[i][j] / -deltaY[i][j];
mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];

}
}
}

else
{
if (deltaY[i][j] >= 0) // dx < 0, dy >= 0
{
if ((deltaX[i][j] - deltaY[i][j]) >= 0)    // direction 3 (SSW)
{
alpha = (float)-deltaX[i][j] / deltaY[i][j];
mag1 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j - 1];
mag2 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j + 1];
}
else                                // direction 4 (SWW)
{
alpha = (float)deltaY[i][j] / -deltaX[i][j];
mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i + 1][j - 1];
mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i - 1][j + 1];
}
}

else // dx < 0, dy < 0
{
if ((-deltaX[i][j] + deltaY[i][j]) >= 0)   // direction 5 (NWW)
{
alpha = (float)deltaY[i][j] / deltaX[i][j];
mag1 = (1 - alpha)*mag[i][j - 1] + alpha*mag[i - 1][j - 1];
mag2 = (1 - alpha)*mag[i][j + 1] + alpha*mag[i + 1][j + 1];
}
else                                // direction 6 (NNW)
{
alpha = (float)deltaX[i][j] / deltaY[i][j];
mag1 = (1 - alpha)*mag[i - 1][j] + alpha*mag[i - 1][j - 1];
mag2 = (1 - alpha)*mag[i + 1][j] + alpha*mag[i + 1][j + 1];
}
}
}

// non-maximal suppression
// compare mag1, mag2 and mag[t]
// if mag[t] is smaller than one of the neighbours then suppress it
if ((mag[i][j] < mag1) || (mag[i][j] < mag2))
newImage[i][j] = 0;//SUPRRESSED
else
{
newImage[i][j] = mag[i][j];
}

}
}
}
}
__global__
void apply_hysteresis(unsigned char **out_pixels, unsigned char **in_pixels, unsigned  char t_high, unsigned  char t_low, int width, int height)
{
/* skip first and last rows and columns, since we'll check them as surrounding neighbors of
* the adjacent rows and columns */

for (unsigned i = 1; i < height - 1; i++) {
for (unsigned j = 1; j < width - 1; j++) {
/* if our input is above the high threshold and the output hasn't already marked it as an edge */
if (out_pixels[i][j] != 0xFF) {
if (in_pixels[i][j] > t_high) {
/* mark as strong edge */
out_pixels[i][j] = 0xff;

/* check 8 immediately surrounding neighbors
* if any of the neighbors are above the low threshold, preserve edge */
trace_immed_neighbors(out_pixels, in_pixels, i, j, t_low);
}
else {
out_pixels[i][j] = 0x00;
}
}
}
}
}
__device__
void trace_immed_neighbors(unsigned char **out_pixels, unsigned char **in_pixels, unsigned i, unsigned j, unsigned char t_low)
{

unsigned char m_edge = 255;
/* directions representing indices of neighbors */

if (((unsigned char)in_pixels[i - 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j - 1] != m_edge)) {
out_pixels[i - 1][j - 1] = m_edge;
}
if (((unsigned char)in_pixels[i - 1][j] >= t_low) && ((unsigned char)out_pixels[i - 1][j] != m_edge)) {
out_pixels[i - 1][j] = m_edge;
}
if (((unsigned char)in_pixels[i - 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i - 1][j + 1] != m_edge)) {
out_pixels[i - 1][j + 1] = m_edge;
}
if (((unsigned char)in_pixels[i][j - 1] >= t_low) && ((unsigned char)out_pixels[i][j - 1] != m_edge)) {
out_pixels[i][j - 1] = m_edge;
}
if (((unsigned char)in_pixels[i][j + 1] >= t_low) && ((unsigned char)out_pixels[i][j + 1] != m_edge)) {
out_pixels[i][j + 1] = m_edge;
}
if (((unsigned char)in_pixels[i + 1][j - 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j - 1] != m_edge)) {
out_pixels[i + 1][j - 1] = m_edge;
}
if (((unsigned char)in_pixels[i + 1][j] >= t_low) && ((unsigned char)out_pixels[i + 1][j] != m_edge)) {
out_pixels[i + 1][j] = m_edge;
}
if (((unsigned char)in_pixels[i + 1][j + 1] >= t_low) && ((unsigned char)out_pixels[i + 1][j + 1] != m_edge)) {
out_pixels[i + 1][j + 1] = m_edge;
}
}

void edge_detector(unsigned char** h_newImg, unsigned char** h_imgbuff, const int WIDTH, const int HEIGHT) {

/* initialize timer */
//struct stopwatch_t* timer = NULL;
//long double t_gaussian, t_gradient, t_nms, t_thres;
//stopwatch_init();
//timer = stopwatch_create();

unsigned char *d_imgbuff;

unsigned char *d_newImage;

cout << "cudaMalloc2d" << endl;




size_t pitch1;
hipError_t err = hipMallocPitch((void**)&d_imgbuff, &pitch1, WIDTH* sizeof(unsigned char), HEIGHT);
if (err == 0)    cout << "cuda2D d_imgbuff finish" << endl;
else cout << "Error :" << err << endl;
size_t pitch2;
err = hipMallocPitch((void**)&d_newImage, &pitch2, WIDTH* sizeof(unsigned char), HEIGHT);
if (err == 0)    cout << "cuda2D d_newImg finish" << endl;
else cout << "Error :" << err << endl;

err = hipMemcpy2D(d_imgbuff, pitch1, h_imgbuff, WIDTH* sizeof(unsigned char), sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyHostToDevice);
if (err != hipSuccess) cout << "Error :" << err << endl;



cout << "hipMalloc finished" << endl;


/*apply gaussian filter*/
cout << "enter gaussian filter" << endl;
dim3 threadsPerBlock(16, 16);
dim3 numBlocks (HEIGHT/threadsPerBlock.x, WIDTH/threadsPerBlock.y);
//stopwatch_start(timer);
gaussian_filter << <numBlocks, threadsPerBlock >> >(d_newImage, d_imgbuff, WIDTH, HEIGHT, pitch1);
//t_gaussian = stopwatch_stop(timer);

//MEMCOPY BACK TO HOST
err = hipMemcpy2D(h_newImg, WIDTH* sizeof(unsigned char), d_newImage, pitch2, sizeof(unsigned char) *WIDTH, HEIGHT, hipMemcpyDeviceToHost);
if (err != hipSuccess) cout << "Error :" << err << endl;


cout << "MEMCOPY BACK TO HOST finished" << endl;

//free device mem

cout << "Free cuda" << endl;
for (int i = 0; i < WIDTH; i++)
{
hipFree(d_imgbuff + i);
hipFree(d_newImage + i);
}
hipFree(d_newImage);
hipFree(d_imgbuff);

//cout << "Time to execute gaussian:" << t_gaussian << endl;
cout << "finished." << endl;


}


